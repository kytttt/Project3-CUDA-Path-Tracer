#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256 
#endif

#ifndef ENABLE_SHARED_MEMORY
#define ENABLE_SHARED_MEMORY 1
#endif


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void UpSweep(int n, int d, int* data) {
            int i = threadIdx.x + blockIdx.x * blockDim.x;
            int step = 1 << (d + 1);
            int count = n / step;
            if (i >= count) return;

            int idx = (i + 1) * step - 1;
            data[idx] += data[idx - (step >> 1)];
        }

        __global__ void DownSweep(int n, int d, int* data) {
            int i = threadIdx.x + blockIdx.x * blockDim.x;
            int step = 1 << (d + 1);
            int count = n / step;
            if (i >= count) return;

            int idx = (i + 1) * step - 1;
            int temp = data[idx - (step >> 1)];
            data[idx - (step >> 1)] = data[idx];
            data[idx] += temp;
        }


        __global__ void shareMemoryScan(int n, const int* idata, int* odata, int* lastElement)
        {
            extern __shared__ int smem[];

            const int i0 = 2 * blockDim.x * blockIdx.x + threadIdx.x;
            const int i1 = 2 * blockDim.x * blockIdx.x + threadIdx.x + blockDim.x;

            smem[threadIdx.x] = (i0 < n) ? idata[i0] : 0;
            smem[threadIdx.x + blockDim.x] = (i1 < n) ? idata[i1] : 0;
            __syncthreads();


            for (int offset = 1; offset < 2 * blockDim.x; offset <<= 1) {
                int idx = ((threadIdx.x + 1) * (offset << 1)) - 1;
                if (idx < 2 * blockDim.x) {
                    smem[idx] += smem[idx - offset];
                }
                __syncthreads();
            }

            int lastEle = smem[2 * blockDim.x - 1];
            if (threadIdx.x == 0) smem[2 * blockDim.x - 1] = 0;
            __syncthreads();


            for (int offset = blockDim.x; offset >= 1; offset >>= 1) {
                int idx = ((threadIdx.x + 1) * (offset << 1)) - 1;
                if (idx < 2 * blockDim.x) {
                    int tmp = smem[idx - offset];
                    smem[idx - offset] = smem[idx];
                    smem[idx] += tmp;
                }
                __syncthreads();
            }

            if (i0 < n) odata[i0] = smem[threadIdx.x];
            if (i1 < n) odata[i1] = smem[threadIdx.x + blockDim.x];


            if (threadIdx.x == 0) lastElement[blockIdx.x] = lastEle;
        }

        __global__ void addLastEle(int n, int* odata, const int* lastElement)
        {

            int add_last = lastElement[blockIdx.x];

            int i0 = 2 * blockDim.x * blockIdx.x + threadIdx.x;
            int i1 = 2 * blockDim.x * blockIdx.x + threadIdx.x + blockDim.x;

            if (i0 < n) odata[i0] += add_last;
            if (i1 < n) odata[i1] += add_last;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();
            // TODO
            if (!ENABLE_SHARED_MEMORY || n <= 2 * BLOCK_SIZE) {

                const int nPow2 = 1 << ilog2ceil(n);

                int* dev_data = nullptr;

                hipMalloc(&dev_data, nPow2 * sizeof(int));
                hipMemset(dev_data, 0, nPow2 * sizeof(int));
                hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

                /*timer().startGpuTimer();*/

                for (int d = 0; d < ilog2ceil(nPow2); ++d) {
                    int count = nPow2 >> (d + 1);
                    if (count == 0) break;
                    int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                    UpSweep <<<gridSize, BLOCK_SIZE >>> (nPow2, d, dev_data);
                    checkCUDAError("UpSweep fail");
                }

                hipMemset(dev_data + (nPow2 - 1), 0, sizeof(int));

                for (int d = ilog2ceil(nPow2) - 1; d >= 0; --d) {
                    int count = nPow2 >> (d + 1);
                    if (count == 0) continue;
                    int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                    DownSweep <<<gridSize, BLOCK_SIZE >>> (nPow2, d, dev_data);
                    checkCUDAError("DownSweep fail");
                }

               /* timer().endGpuTimer();*/

                hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

                hipFree(dev_data);
            }

            else {

                const int tile = 2 * BLOCK_SIZE;
                const int numBlocks = (n + tile - 1) / tile;

                int* dev_in = nullptr, * dev_out = nullptr;
                int* dev_lastEle = nullptr;

                hipMalloc(&dev_in, n * sizeof(int));
                hipMalloc(&dev_out, n * sizeof(int));
                hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

                hipMalloc(&dev_lastEle, numBlocks * sizeof(int));


                timer().startGpuTimer();

                shareMemoryScan <<<numBlocks, BLOCK_SIZE, tile * sizeof(int) >>> (n, dev_in, dev_out, dev_lastEle);
                

                int m = 1 << ilog2ceil(numBlocks);

                int* dev_tmp = nullptr;
                hipMalloc(&dev_tmp, m * sizeof(int));
                hipMemset(dev_tmp, 0, m * sizeof(int));
                hipMemcpy(dev_tmp, dev_lastEle, numBlocks * sizeof(int), hipMemcpyDeviceToDevice);

                for (int d = 0; d < ilog2ceil(m); ++d) {
                    int count = m >> (d + 1);
                    if (count == 0) break;
                    int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                    UpSweep <<<gridSize, BLOCK_SIZE >> > (m, d, dev_tmp);
                }

                hipMemset(dev_tmp + (m - 1), 0, sizeof(int));

                for (int d = ilog2ceil(m) - 1; d >= 0; --d) {
                    int count = m >> (d + 1);
                    if (count == 0) continue;
                    int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                    DownSweep <<<gridSize, BLOCK_SIZE >>> (m, d, dev_tmp);
                }

                hipMemcpy(dev_lastEle, dev_tmp, numBlocks * sizeof(int), hipMemcpyDeviceToDevice);
                hipFree(dev_tmp);

                addLastEle <<<numBlocks, BLOCK_SIZE >>> (n, dev_out, dev_lastEle);

                timer().endGpuTimer();

                hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);

                hipFree(dev_in);
                hipFree(dev_out);
                hipFree(dev_lastEle);
            }


        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // TODO
			/*const int blockSize = 256;*/
			const int gridSizeN = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

			int* boolArray = nullptr;
			int* indices = nullptr;
			int* dev_data = nullptr;
            int* output = nullptr;

			const int nPow2 = 1 << ilog2ceil(n);

			hipMalloc(&boolArray, n * sizeof(int));
			hipMalloc(&indices, nPow2 * sizeof(int));
			hipMalloc(&dev_data, n * sizeof(int));
            hipMalloc(&output, n * sizeof(int));

            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

			Common::kernMapToBoolean <<<gridSizeN, BLOCK_SIZE >>> (n, boolArray, dev_data);
			checkCUDAError("ToBoolean fail");

			hipMemset(indices, 0, nPow2 * sizeof(int));
			hipMemcpy(indices, boolArray, n * sizeof(int), hipMemcpyDeviceToDevice);

            //upsweep
            for (int d = 0; d < ilog2ceil(nPow2); ++d) {
                int count = nPow2 >> (d + 1);
                if (count == 0) break;
                int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                UpSweep <<<gridSize, BLOCK_SIZE >>> (nPow2, d, indices);
                checkCUDAError("UpSweep fail");
            }

            hipMemset(indices + (nPow2 - 1), 0, sizeof(int));

            //downsweep
            for (int d = ilog2ceil(nPow2) - 1; d >= 0; --d) {
                int count = nPow2 >> (d + 1);
                if (count == 0) continue;
                int gridSize = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;
                DownSweep <<<gridSize, BLOCK_SIZE >>> (nPow2, d, indices);
                checkCUDAError("DownSweep fail");
            }

            int lastIndex = 0, lastBool = 0;
            hipMemcpy(&lastIndex, indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastBool, boolArray + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            int count = lastIndex + lastBool;

            //scatter
            Common::kernScatter <<<gridSizeN, BLOCK_SIZE >>> (
                n, output, dev_data, boolArray, indices);
            checkCUDAError("Scatter fail");

            timer().endGpuTimer();

            hipMemcpy(odata, output, count * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_data);
            hipFree(output);
            hipFree(boolArray);
            hipFree(indices);


            return count;
            return -1;
        }
    }
}
